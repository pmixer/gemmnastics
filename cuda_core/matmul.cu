#include <ctime>

#include <iostream>
#include <iomanip>

#include <assert.h>

#include <hip/hip_runtime.h>

// inner product based matmul
template <typename val_t, typename pos_t>
__global__ void K0(const val_t *A, const val_t *B, val_t *C,
                   const pos_t M, const pos_t K, const pos_t N)
{
    for (pos_t m = blockIdx.x * blockDim.x + threadIdx.x; m < M; m += gridDim.x * blockDim.x)
    {
        for (pos_t n = blockIdx.y * blockDim.y + threadIdx.y; n < N; n += gridDim.y * blockDim.y)
        {
            val_t accum = 0;
            for (pos_t k = 0; k < K; k++)
            {
                accum += A[m * K + k] * B[k * N + n];
            }
            C[m * N + n] = accum;
        }
    }
}

template <typename val_t, typename pos_t>
inline void K0_launcher(const val_t *A, const val_t *B, val_t *C,
                        const pos_t M, const pos_t K, const pos_t N)
{
    dim3 gd(1, 1, 1);
    dim3 bd(32, 32, 1);
    K0<<<gd, bd>>>(A, B, C, M, K, N);
}

// https://github.com/BryanCatanzaro/inplace
// sth wrong as in below, inplace transpose harder than I expected
template <typename val_t, typename pos_t, pos_t TM, pos_t TN>
__global__ void ITK(val_t *mat, const pos_t M, const pos_t N)
{
    assert(blockDim.x <= TM);
    assert(blockDim.y <= TN);
    // assert(blockDim.x == blockDim.y);
    __shared__ val_t tile[2][TM][TN];
    // bank conflicit as it will access elems by rows and cols, cache on registers?
    for (pos_t m = blockIdx.x * blockDim.x + threadIdx.x; m < M; m += gridDim.x * blockDim.x)
    {
        for (pos_t n = blockIdx.y * blockDim.y + threadIdx.y; n < N; n += gridDim.y * blockDim.y)
        {
            if ((m > n) && (M >= N) || (m < n) && (M < N))
            {
                tile[0][n % blockDim.y][m % blockDim.x] = mat[m * N + n];
                tile[1][m % blockDim.x][n % blockDim.y] = mat[n * M + m];
            }
            __syncthreads();
            if ((m > n) && (M >= N) || (m < n) && (M < N))
            {
                mat[n * M + m] = tile[0][m % blockDim.x][n % blockDim.y];
                mat[m * N + n] = tile[1][m % blockDim.x][n % blockDim.y];
            }
        }
    }
}

template <typename val_t, typename pos_t, pos_t TM, pos_t TN>
__global__ void TK(val_t *original, const pos_t M, const pos_t N, val_t *transposed)
{
    assert(blockDim.x <= TM);
    assert(blockDim.y <= TN);
    __shared__ val_t tile[TM][TN];
    // bank conflicit as it will access elems by rows and cols, cache on registers?
    for (pos_t m = blockIdx.x * blockDim.x + threadIdx.x; m < M; m += gridDim.x * blockDim.x)
    {
        for (pos_t n = blockIdx.y * blockDim.y + threadIdx.y; n < N; n += gridDim.y * blockDim.y)
        {
            tile[m % blockDim.x][n % blockDim.y] = original[m * N + n];
            __syncthreads();
            transposed[n * M + m] = tile[m % blockDim.x][n % blockDim.y];
        }
    }
}

template <typename val_t, typename pos_t>
void transpose(val_t *original,  const pos_t nr, const pos_t nc, bool inplace=true, val_t *transposed=nullptr)
{
    dim3 gd(6, 8);
    dim3 bd(32, 32);
    assert(inplace == false); // as not yet finished
    // ITK<val_t, pos_t, 33, 33><<<gd, bd>>>(original, nr, nc);

    TK<val_t, pos_t, 33, 33><<<gd, bd>>>(original, nr, nc, transposed);
}

// outer product based matmul
template <typename val_t, typename pos_t, pos_t MAX_M, pos_t MAX_N>
__global__ void K1(const val_t *A, const val_t *B, val_t *C,
                   const pos_t M, const pos_t K, const pos_t N)
{
    __shared__ val_t AC[MAX_M], BR[MAX_N]; // A column, B row, A better transposed but not yet
    assert(M <= MAX_M); // get to re-balance MAX_M and MAX_N if one of them too large
    assert(N <= MAX_N); // get to add another 2D loop if both MAX_M + MAX_N too large
    for (int k = blockIdx.x; k < K; k += gridDim.x)
    {
        for (pos_t m = threadIdx.x; m < M; m += blockDim.x)
        {
            AC[m] = A[k * M + m]; // if A is column-major
            // AC[m] = A[m * K + k]; // if A is row-major
        }

        for (pos_t n = threadIdx.x; n < N; n += blockDim.x)
        {
            BR[n] = B[k * N + n];
        }

        __syncthreads();

        for (pos_t i = threadIdx.x; i < M * N; i += blockDim.x)
        {
            pos_t m = i / N, n = i % N;
            val_t v = AC[m] *  BR[n];
            atomicAdd(C + m * N  + n, v);
        }
    }
}

template <typename val_t, typename pos_t>
inline void K1_launcher(val_t *A, const val_t *B, val_t *C,
                        const pos_t M, const pos_t K, const pos_t N)
{
    hipMemset(C, 0, M * N * sizeof(val_t)); // w/o memset also w/ correct res?
    K1<val_t, pos_t, 5000, 5000><<<K, 1024>>>(A, B, C, M, K, N);
}

#include <stdint.h>

#define BS 128
#define KS 16

// vanilla tile based matmul, recursively divide and conquer, too many loops
__global__ void K2(const int *A, const int *B, int *C,
                   const uint32_t M, const uint32_t K, const uint32_t N)
{
    for (uint32_t m0 = blockIdx.x * BS; m0 < M; m0 += gridDim.x * BS)
    {
        for (uint32_t n0 = blockIdx.y * BS; n0 < N; n0 += gridDim.y * BS)
        {
            // register int4 CT[2][2][4] = {make_int4(0, 0, 0, 0)};
            register int CT[2][2][4][4] = {0};

            // how to share shm content cross blocks?
            for (uint32_t k0 = 0; k0 < K; k0 += KS) // KS = 16 = blockDim.x .y
            {
                __shared__ int AT[KS][BS];

                for (uint32_t i = threadIdx.x; i < BS; i += blockDim.x)
                {
                    for (uint32_t j = threadIdx.y; j < KS; j += blockDim.y)
                    {
                        uint32_t m = m0 + i, k = k0 + j;
                        if (m < M && k < K)
                        {
                            AT[j][i] = A[m * K + k];
                        }
                        else
                        {
                            AT[j][i] = 0;
                        }
                    }
                }

                __shared__ int BT[KS][BS];

                for (uint32_t i = threadIdx.x; i < KS; i += blockDim.x)
                {
                    for (uint32_t j = threadIdx.y; j < BS; j += blockDim.y)
                    {
                        uint32_t k = k0 + i, n = n0 + j;
                        if (k < K && n < N)
                        {
                            BT[i][j] = B[k * N + n];
                        }
                        else
                        {
                            BT[i][j] = 0;
                        }
                    }
                }

                register int4 AC[2], BR[2];

                #pragma unroll
                for (int tk = 0; tk < KS; tk++)
                {
                    // will make_uint4 trigger some smart compiler behaviour to avoid explicit use of int4 dtype here?
                    AC[0] = make_int4(AT[tk][threadIdx.x*4], AT[tk][threadIdx.x*4+1], AT[tk][threadIdx.x*4+2], AT[tk][threadIdx.x*4+3]);
                    AC[1] = make_int4(AT[tk][(blockDim.x+threadIdx.x)*4], AT[tk][(blockDim.x+threadIdx.x)*4+1], AT[tk][(blockDim.x+threadIdx.x)*4+2], AT[tk][(blockDim.x+threadIdx.x)*4+3]);
                    BR[0] = make_int4(BT[tk][threadIdx.y*4], BT[tk][threadIdx.y*4+1], BT[tk][threadIdx.y*4+2], BT[tk][threadIdx.y*4+3]);
                    BR[1] = make_int4(BT[tk][(blockDim.y+threadIdx.y)*4], BT[tk][(blockDim.y+threadIdx.y)*4+1], BT[tk][(blockDim.y+threadIdx.y)*4+2], BT[tk][(blockDim.y+threadIdx.y)*4+3]);


                    CT[0][0][0][0] += AC[0].x * BR[0].x;
                    CT[0][0][0][1] += AC[0].x * BR[0].y;
                    CT[0][0][0][2] += AC[0].x * BR[0].z;
                    CT[0][0][0][3] += AC[0].x * BR[0].w;
                    CT[0][0][1][0] += AC[0].y * BR[0].x;
                    CT[0][0][1][1] += AC[0].y * BR[0].y;
                    CT[0][0][1][2] += AC[0].y * BR[0].z;
                    CT[0][0][1][3] += AC[0].y * BR[0].w;
                    CT[0][0][2][0] += AC[0].z * BR[0].x;
                    CT[0][0][2][1] += AC[0].z * BR[0].y;
                    CT[0][0][2][2] += AC[0].z * BR[0].z;
                    CT[0][0][2][3] += AC[0].z * BR[0].w;
                    CT[0][0][3][0] += AC[0].w * BR[0].x;
                    CT[0][0][3][1] += AC[0].w * BR[0].y;
                    CT[0][0][3][2] += AC[0].w * BR[0].z;
                    CT[0][0][3][3] += AC[0].w * BR[0].w;

                    CT[0][1][0][0] += AC[0].x * BR[1].x;
                    CT[0][1][0][1] += AC[0].x * BR[1].y;
                    CT[0][1][0][2] += AC[0].x * BR[1].z;
                    CT[0][1][0][3] += AC[0].x * BR[1].w;
                    CT[0][1][1][0] += AC[0].y * BR[1].x;
                    CT[0][1][1][1] += AC[0].y * BR[1].y;
                    CT[0][1][1][2] += AC[0].y * BR[1].z;
                    CT[0][1][1][3] += AC[0].y * BR[1].w;
                    CT[0][1][2][0] += AC[0].z * BR[1].x;
                    CT[0][1][2][1] += AC[0].z * BR[1].y;
                    CT[0][1][2][2] += AC[0].z * BR[1].z;
                    CT[0][1][2][3] += AC[0].z * BR[1].w;
                    CT[0][1][3][0] += AC[0].w * BR[1].x;
                    CT[0][1][3][1] += AC[0].w * BR[1].y;
                    CT[0][1][3][2] += AC[0].w * BR[1].z;
                    CT[0][1][3][3] += AC[0].w * BR[1].w;

                    CT[1][0][0][0] += AC[1].x * BR[0].x;
                    CT[1][0][0][1] += AC[1].x * BR[0].y;
                    CT[1][0][0][2] += AC[1].x * BR[0].z;
                    CT[1][0][0][3] += AC[1].x * BR[0].w;
                    CT[1][0][1][0] += AC[1].y * BR[0].x;
                    CT[1][0][1][1] += AC[1].y * BR[0].y;
                    CT[1][0][1][2] += AC[1].y * BR[0].z;
                    CT[1][0][1][3] += AC[1].y * BR[0].w;
                    CT[1][0][2][0] += AC[1].z * BR[0].x;
                    CT[1][0][2][1] += AC[1].z * BR[0].y;
                    CT[1][0][2][2] += AC[1].z * BR[0].z;
                    CT[1][0][2][3] += AC[1].z * BR[0].w;
                    CT[1][0][3][0] += AC[1].w * BR[0].x;
                    CT[1][0][3][1] += AC[1].w * BR[0].y;
                    CT[1][0][3][2] += AC[1].w * BR[0].z;
                    CT[1][0][3][3] += AC[1].w * BR[0].w;

                    CT[1][1][0][0] += AC[1].x * BR[1].x;
                    CT[1][1][0][1] += AC[1].x * BR[1].y;
                    CT[1][1][0][2] += AC[1].x * BR[1].z;
                    CT[1][1][0][3] += AC[1].x * BR[1].w;
                    CT[1][1][1][0] += AC[1].y * BR[1].x;
                    CT[1][1][1][1] += AC[1].y * BR[1].y;
                    CT[1][1][1][2] += AC[1].y * BR[1].z;
                    CT[1][1][1][3] += AC[1].y * BR[1].w;
                    CT[1][1][2][0] += AC[1].z * BR[1].x;
                    CT[1][1][2][1] += AC[1].z * BR[1].y;
                    CT[1][1][2][2] += AC[1].z * BR[1].z;
                    CT[1][1][2][3] += AC[1].z * BR[1].w;
                    CT[1][1][3][0] += AC[1].w * BR[1].x;
                    CT[1][1][3][1] += AC[1].w * BR[1].y;
                    CT[1][1][3][2] += AC[1].w * BR[1].z;
                    CT[1][1][3][3] += AC[1].w * BR[1].w;
                }
            }

            // if ((m0 + threadIdx.x * 4) < M && n0 + threadIdx.y * 4 < N)
            //     ((int4 *)C)[((m0 + threadIdx.x * 4) * N + n0 + threadIdx.y * 4)] = CT[0][0][0];
            // ((int4 *)C)[0] = CT[0][0][0]; // this line works for write mockup, but the above failed

            #pragma unroll // so backoff to w/o packed int4 write
            for (int i = 0; i < 2; i++)
            {
                for (int j = 0; j < 2; j++)
                {
                    for (int k = 0; k < 4; k++)
                    {
                        for (int v = 0; v < 4; v++)
                        {
                            uint32_t m = m0 + threadIdx.x * 4 + i * blockDim.x * 4 + k;
                            uint32_t n = n0 + threadIdx.y * 4 + j * blockDim.y * 4 + v;
                            if (m < M && n < N)
                            {
                                C[m * N + n] = CT[i][j][k][v];
                            }
                        }
                    }
                }
            }
            // 4 vals packed is kind of troublesome in programming
        }
    }
}

inline void K2_launcher(const int *A, const int *B, int *C,
                        const uint32_t M, const uint32_t K, const uint32_t N)
{
    dim3 gd(6 * 3, 8 * 2, 1);
    dim3 bd(16, 16, 1); // square tile? more threads? shared mem for larger block?
    K2<<<gd, bd>>>(A, B, C, M, K, N);
}

// tall and thin matmul

// tiny matmul

// for ffn?

// loop over a wide range of M, N, K, dtype, CC/TC combinations

// okay...cublas

// what if M, N, K or grid size or block size out of boundary?

// what if we want to decouple the building blocks to variants

// okay...cutlass

template <typename val_t, typename pos_t>
struct Matmul
{
    val_t *A, *B, *C, *refC;
    val_t *dA, *dB, *dC;
    val_t *TA, *dTA;
    pos_t M, K, N;

    hipEvent_t es, ee;
    float t_ms;

    void host_alloc()
    {
        A = (val_t *)malloc(M * K * sizeof(val_t));
        B = (val_t *)malloc(K * N * sizeof(val_t));
        C = (val_t *)malloc(M * N * sizeof(val_t));
        refC = (val_t *)malloc(M * N * sizeof(val_t));
        TA = (val_t *)malloc(M * K * sizeof(val_t));
    }

    void host_free()
    {
        free(A);
        free(B);
        free(C);
        free(refC);
        free(TA);
        A = B = C = refC = TA = nullptr;
    }

    void fill_vals()
    {
        std::srand(std::time({}));
        for (pos_t m = 0; m < M; m++)
        {
            for (pos_t k = 0; k < K; k++)
            {
                A[m * K + k] = std::rand() % 2333;
            }
        }

        for (pos_t k = 0; k < K; k++)
        {
            for (pos_t n = 0; n < N; n++)
            {
                B[k * N + n] = std::rand() % 2333;
            }
        }
    }


    void get_ref()
    {
        for (pos_t m = 0; m < M; m++)
        {
            for (pos_t n = 0; n < N; n++)
            {
                val_t val = 0;
                for (pos_t k = 0; k < K; k++)
                {
                    val += A[m * K + k] * B[k * N + n];
                }
                refC[m * N + n] = val;
            }
        }
    }


    void device_alloc()
    {
        hipMalloc(&dA, M * K * sizeof(val_t));
        hipMalloc(&dB, K * N * sizeof(val_t));
        hipMalloc(&dC, M * N * sizeof(val_t));

        hipMalloc(&dTA, M * K * sizeof(val_t));
    }


    void device_free()
    {
        hipFree(dA);
        hipFree(dB);
        hipFree(dC);

        hipFree(dTA);

        dA = dB = dC = dTA = nullptr;
    }


    void h2d()
    {
        hipMemcpy(dA, A, M * K * sizeof(val_t), hipMemcpyHostToDevice);
        hipMemcpy(dB, B, K * N * sizeof(val_t), hipMemcpyHostToDevice);
    }


    inline void tic(std::string msg)
    {
        std::cout << msg << " ";
        hipEventRecord(es);
    }


    inline void toc()
    {
        hipEventRecord(ee);
        hipEventSynchronize(ee);
        hipEventElapsedTime(&t_ms, es, ee);
        std::cout << "elapsed time: " << t_ms << "ms" << std::endl;
    }

    inline void get_res()
    {
        // K0_launcher(dA, dB, dC, M, K, N);
        // K1_launcher(dTA, dB, dC, M, K, N);
        K2_launcher(dA, dB, dC, M, K, N);
    }

    void d2h()
    {
        hipMemcpy(C, dC, M * N * sizeof(val_t), hipMemcpyDeviceToHost);
        hipMemcpy(TA, dTA, M * K * sizeof(val_t), hipMemcpyDeviceToHost);
    }

    void print_mat(std::string mat_name, val_t *data, pos_t nr, pos_t nc)
    {
        std::cout << mat_name << std::endl; // << std::setw(16);
        for (pos_t ir = 0; ir < nr; ir++)
        {
            for (pos_t ic = 0; ic < nc; ic++)
            {
                std::cout << data[ir * nc + ic] << "\t";
            }
            std::cout << std::endl;
        }
    }

    void print()
    {
        print_mat("A", A, M, K);
        print_mat("B", B, K, N);
        print_mat("refC", refC, M, N);
        print_mat("C", C, M, N);
    }

    bool match()
    {
        bool matched = true;
        for (pos_t i = 0; i < M * N; i++)
        {
            if (C[i] != refC[i])
            {
                std::cout << "mismatch, pos in 1D array: " << i << std::endl;
                matched = false;
                break;
            }
        }
        return matched;
    }

    void compute()
    {
        tic("gpu matmul");
        get_res();
        toc();
        d2h();
        assert(match());
        // std::cout << "correct? " << (match()?"yes":"no") << std::endl;
    }

    Matmul() = delete;

    Matmul(pos_t m, pos_t k, pos_t n) : M(m), K(k), N(n)
    {
        host_alloc();
        fill_vals();
        get_ref();

        device_alloc();
        hipEventCreate(&es);
        hipEventCreate(&ee);

        h2d();
        transpose(dA, M, K, false, dTA);
    }


    ~Matmul()
    {
        host_free();
        device_free();
        hipEventDestroy(es);
        hipEventDestroy(ee);
    }
};

__global__ void fooKernel()
{
}

template <typename val_t, typename pos_t>
void transpose_test(Matmul<val_t, pos_t> &mm)
{
    // mm.print_mat("original A", mm.A, 51, 31);
    // mm.d2h();
    // mm.print_mat("transposed A", mm.TA, 31, 51);
    // mm.print();
}

int main()
{
    fooKernel<<<1, 1>>>();

    // Matmul<int, uint32_t> mm(511 * 4, 513 * 4, 519 * 4);
    // Matmul<int, uint32_t> mm(32, 32, 24);
    // mm.compute(); // comp refC by cublas?
    // mm.print();


    for (int m = 1; m < 256; m++)
    {
        for (int k = 1; k < 188; k++)
        {
            for (int n = 1; n < 300; n++)
            {
                std::cout << "m=" << m*4 << ",k=" << k*4 << ",n=" << n*4 << std::endl;
                Matmul<int, uint32_t> mm(m*4, k*4, n*4);
                mm.compute();
            }
        }
    }
    return 0;
}